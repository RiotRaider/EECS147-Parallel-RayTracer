#include "hip/hip_runtime.h"
// Student Name: Justin Sanders
// Student ID: 862192429

#include <stdio.h>
#include <stdlib.h>

#include "render_world.h"
#include "flat_shader.h"
#include "object.cuh"
#include "light.h"
#include "ray.cuh"

#include "support.h"
#include "kernel.cuh"

extern bool enable_acceleration;

Render_World::~Render_World()
{
    
    for (auto a : all_objects)
        delete a;
    for (auto a : all_shaders)
        delete a;
    for (auto a : all_colors)
        delete a;
    for (auto a : lights)
        delete a;
    
        
}

// Find and return the Hit structure for the closest intersection.  Be careful
// to ensure that hit.dist>=small_t.
std::pair<Shaded_Object, Hit> Render_World::Closest_Intersection(const Ray &ray) const
{
    double min_t = std::numeric_limits<double>::max();
    Shaded_Object o;
    Hit h;
    std::pair<Shaded_Object, Hit> obj = {o, h};
    Hit hit_test;
    for (auto a : this->objects)
    {
        hit_test = a.object->Intersection(ray, -1);
        if (hit_test.dist >= small_t)
        {
            if (hit_test.dist < min_t)
            {
                min_t = hit_test.dist;
                obj.first = a;
                obj.second = hit_test;
            }
        }
    }
    return obj;
}

// set up the initial view ray and call
void Render_World::Render_Pixel(const ivec2 &pixel_index)
{
    // set up the initial view ray here
    vec3 rayDir = (camera.World_Position(pixel_index) - camera.position).normalized();
    Ray ray(camera.position, rayDir);
    vec3 color = Cast_Ray(ray, 1);
    camera.Set_Pixel(pixel_index, Pixel_Color(color));
}

void Render_World::Render()
{
    Timer timer;

    if (gpu_on) {
        //compute on gpu
        printf("Render image on gpu..."); fflush(stdout);
        startTime(&timer);

        //launch kernel
        //temporary - test launch kernel with vec class

        /*================================*/

        //Hit
        
        Hit *e = new Hit;
        Hit *f = new Hit;
        
        for (int i = 0; i < 2; i++) {
            e->uv[i] = 10;
        }

        e->dist = 100;
        e->triangle = 5;

        for (int i = 0; i < 2; i++) {
            f->uv[i] = 20;
        }

        f->dist = 200;
        f->triangle = 10;

        printf("\nHit:\nOn host (print) e: uv=(%.2f, %.2f), dist=%.2f, triangle=%d\n", e->uv[0], e->uv[1], e->dist, e->triangle);
        printf("On host (print) f: uv=(%.2f, %.2f), dist=%.2f, triangle=%d\n", f->uv[0], f->uv[1], f->dist, f->triangle);

        //add
        e->uv += f->uv;
        printf("On host (after e + f): uv=(%.2f, %.2f), dist=%.2f, triangle=%d\n", e->uv[0], e->uv[1], e->dist, e->triangle);
        
        launch_by_pointer_hit(e, f);
        printf("On host (after by-pointer): uv=(%.2f, %.2f), dist=%.2f, triangle=%d\n", e->uv[0], e->uv[1], e->dist, e->triangle);

        launch_by_ref_hit(*e, *f);
        printf("On host (after by-ref): uv=(%.2f, %.2f), dist=%.2f, triangle=%d\n", e->uv[0], e->uv[1], e->dist, e->triangle);

        launch_by_value_hit(*e, *f);
        printf("On host (after by-value): uv=(%.2f, %.2f), dist=%.2f, triangle=%d\n", e->uv[0], e->uv[1], e->dist, e->triangle);
        

        //Ray
        Ray *q = new Ray;
        //Ray *r = new Ray;
        
        q->endpoint = {10, 20, 30};
        q->direction = {0.10, 0.20, 0.30};
        
        vec3 q_ray_point = q->Point(5);

        printf("\nRay:\nOn host (print) q: endpoint=(%.2f, %.2f, %.2f), direction=(%.2f, %.2f, %.2f), point=(%.2f, %.2f, %.2f)\n", 
            q->endpoint[0], q->endpoint[1], q->endpoint[2], q->direction[0], q->direction[1], q->direction[2], q_ray_point[0], q_ray_point[1], q_ray_point[2]);
        
        
        launch_by_pointer_ray(q);
        printf("On host (after by-pointer) q: endpoint=(%.2f, %.2f, %.2f), direction=(%.2f, %.2f, %.2f), point=(%.2f, %.2f, %.2f)\n", 
            q->endpoint[0], q->endpoint[1], q->endpoint[2], q->direction[0], q->direction[1], q->direction[2], q_ray_point[0], q_ray_point[1], q_ray_point[2]);
        
        launch_by_ref_ray(*q);
        printf("On host (after by-ref) q: endpoint=(%.2f, %.2f, %.2f), direction=(%.2f, %.2f, %.2f), point=(%.2f, %.2f, %.2f)\n", 
            q->endpoint[0], q->endpoint[1], q->endpoint[2], q->direction[0], q->direction[1], q->direction[2], q_ray_point[0], q_ray_point[1], q_ray_point[2]);

        launch_by_value_ray(*q);
        printf("On host (after by-val) q: endpoint=(%.2f, %.2f, %.2f), direction=(%.2f, %.2f, %.2f), point=(%.2f, %.2f, %.2f)\n", 
            q->endpoint[0], q->endpoint[1], q->endpoint[2], q->direction[0], q->direction[1], q->direction[2], q_ray_point[0], q_ray_point[1], q_ray_point[2]);
        
        //delete these pointers
        delete e;
        delete f;
        delete q;

        hipDeviceReset();

        /*================================*/

        stopTime(&timer); 
        printf("\n...%f s\n", elapsedTime(timer));
    }
    else {
        //compute on cpu
        printf("Render image on cpu..."); fflush(stdout);
        startTime(&timer);

        for (int j = 0; j < camera.number_pixels[1]; j++) {
            for (int i = 0; i < camera.number_pixels[0]; i++) {
                Render_Pixel(ivec2(i, j));
            }
        }

        stopTime(&timer); 
        printf("%f s\n", elapsedTime(timer));
    }
}

// cast ray and return the color of the closest intersected surface point,
// or the background color if there is no object intersection
vec3 Render_World::Cast_Ray(const Ray &ray, int recursion_depth) const
{
    vec3 color;
    if (recursion_depth > recursion_depth_limit)
    {
        color.make_zero();
        return color;
    }
    // Set color to background color as default
    Hit dummyHit;
    
    
    // determine the color here (change if not at recursion limit)
    std::pair<Shaded_Object, Hit> obj = Closest_Intersection(ray);
    if (obj.first.object != nullptr)
    {
        vec3 q = ray.endpoint + (ray.direction * obj.second.dist);
        vec3 n = (obj.first.object->Normal(ray, obj.second)).normalized();
        color = obj.first.shader->Shade_Surface(*this, ray, obj.second, q, n, recursion_depth);
    }else{
        if (background_shader == nullptr)
        {
            color.make_zero();
        }
        else
        {
            color = background_shader->Shade_Surface(*this, ray, dummyHit, ray.direction, ray.direction, 1);
        }
    }

    return color;
}
