#include "point_light.cuh"
#include "parse.h"
#include "color.h"

Point_Light::Point_Light(const Point_Light& l)
{
    position = l.position;
    hipMallocManaged(&color, sizeof(Color));
    color = l.color;
}
Point_Light::Point_Light(const Parse* parse,std::istream& in)
{
    in>>name>>position;
    color=parse->Get_Color(in);
    in>>brightness;
}

vec3 Point_Light::Emitted_Light(const vec3& vector_to_light) const
{
    return color->Get_Color({})*brightness/(4*pi*vector_to_light.magnitude_squared());
}
